#include "hip/hip_runtime.h"
// cuda_topk_rankk.cu
// Rowwise TopK kernels: warp-heap / warp-bitonic (float32).
// Compile to PTX and load via cust/cudarc. K up to 1024 per row in single pass (extend for larger).

#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

extern "C" {

constexpr int WARP_LANES = 32;
constexpr int BLOCK_WARPS = 4;
constexpr int THREADS_PER_BLOCK = WARP_LANES * BLOCK_WARPS;
constexpr int KEEP_PER_THREAD = 8;
static_assert(THREADS_PER_BLOCK % WARP_LANES == 0, "blockDim.x must be warp-aligned");
static_assert(BLOCK_WARPS * WARP_LANES == THREADS_PER_BLOCK, "block warp geometry mismatch");

__device__ __forceinline__ int linear_row_index() {
  long long gx = static_cast<long long>(gridDim.x);
  long long gy = static_cast<long long>(gridDim.y);
  long long x = static_cast<long long>(blockIdx.x);
  long long y = static_cast<long long>(blockIdx.y);
  long long z = static_cast<long long>(blockIdx.z);
  long long row = x + y * gx + z * gx * gy;
  return static_cast<int>(row);
}

struct HeapEntry {
  float value;
  int column;
  int slot;
  int tid;
};

template <typename Comparator>
__device__ __forceinline__ bool prefer_entry(
    const HeapEntry& candidate,
    const HeapEntry& current,
    Comparator cmp) {
  if (candidate.column < 0) {
    return false;
  }
  if (current.column < 0) {
    return true;
  }
  if (cmp(candidate.value, current.value)) {
    return true;
  }
  if (cmp(current.value, candidate.value)) {
    return false;
  }
  if (candidate.column < current.column) {
    return true;
  }
  if (candidate.column > current.column) {
    return false;
  }
  if (candidate.tid < current.tid) {
    return true;
  }
  if (candidate.tid > current.tid) {
    return false;
  }
  return candidate.slot < current.slot;
}

template <typename Comparator>
__device__ __forceinline__ HeapEntry reduce_warp(HeapEntry entry, Comparator cmp) {
  unsigned mask = __activemask();
  int lane = threadIdx.x & (WARP_LANES - 1);
  for (int offset = WARP_LANES / 2; offset > 0; offset >>= 1) {
    int src_lane = lane + offset;
    bool other_active = (src_lane < WARP_LANES) && ((mask >> src_lane) & 1u);
    float other_value = __shfl_down_sync(mask, entry.value, offset);
    int other_col = __shfl_down_sync(mask, entry.column, offset);
    int other_slot = __shfl_down_sync(mask, entry.slot, offset);
    int other_tid = __shfl_down_sync(mask, entry.tid, offset);
    HeapEntry other{other_value, other_col, other_slot, other_tid};
    if (other_active && prefer_entry(other, entry, cmp)) {
      entry = other;
    }
  }
  return entry;
}

struct GreaterThan {
  __device__ bool operator()(float lhs, float rhs) const { return lhs > rhs; }
};

struct LessThan {
  __device__ bool operator()(float lhs, float rhs) const { return lhs < rhs; }
};

template <typename Comparator>
struct HeapTraits;

template <>
struct HeapTraits<GreaterThan> {
  static __device__ __forceinline__ float sentinel() { return -HIP_INF_F; }
};

template <>
struct HeapTraits<LessThan> {
  static __device__ __forceinline__ float sentinel() { return HIP_INF_F; }
};

template <typename Comparator>
__device__ __forceinline__ void heap_select_rowwise_kernel_impl(
    const float* __restrict__ X,
    int rows,
    int cols,
    int k,
    float* __restrict__ out_vals,
    int* __restrict__ out_idx,
    float* s_vals,
    int* s_idx,
    HeapEntry* warp_entries,
    HeapEntry* block_choice) {
  int row = linear_row_index();
  if (row >= rows) return;
  int tid = threadIdx.x;
  int stride = blockDim.x;
  if (stride != THREADS_PER_BLOCK) return;

  Comparator cmp;
  float sentinel = HeapTraits<Comparator>::sentinel();

  if (cols <= 0 || k <= 0) {
    if (tid == 0 && k > 0) {
      size_t out_base = static_cast<size_t>(row) * static_cast<size_t>(k);
      for (int oi = 0; oi < k; ++oi) {
        out_vals[out_base + oi] = HIP_NAN_F;
        out_idx[out_base + oi] = -1;
      }
    }
    return;
  }

  float vbuf[KEEP_PER_THREAD];
  int ibuf[KEEP_PER_THREAD];
  #pragma unroll
  for (int i = 0; i < KEEP_PER_THREAD; ++i) {
    vbuf[i] = sentinel;
    ibuf[i] = -1;
  }

  const float* row_ptr = X + static_cast<size_t>(row) * static_cast<size_t>(cols);

  for (int c = tid; c < cols; c += stride) {
    float v = row_ptr[c];
    #pragma unroll
    for (int pos = 0; pos < KEEP_PER_THREAD; ++pos) {
      if (cmp(v, vbuf[pos]) || (v == vbuf[pos] && (ibuf[pos] < 0 || c < ibuf[pos]))) {
        for (int q = KEEP_PER_THREAD - 1; q > pos; --q) {
          vbuf[q] = vbuf[q - 1];
          ibuf[q] = ibuf[q - 1];
        }
        vbuf[pos] = v;
        ibuf[pos] = c;
        break;
      }
    }
  }

  int base = tid * KEEP_PER_THREAD;
  #pragma unroll
  for (int i = 0; i < KEEP_PER_THREAD; ++i) {
    s_vals[base + i] = vbuf[i];
    s_idx[base + i] = ibuf[i];
  }

  int warp = tid / WARP_LANES;
  int lane = tid & (WARP_LANES - 1);

  int take = k < cols ? k : cols;

  for (int oi = 0; oi < take; ++oi) {
    HeapEntry thread_best{sentinel, -1, -1, -1};
    #pragma unroll
    for (int s = 0; s < KEEP_PER_THREAD; ++s) {
      HeapEntry candidate{s_vals[base + s], s_idx[base + s], s, tid};
      if (prefer_entry(candidate, thread_best, cmp)) {
        thread_best = candidate;
      }
    }

    HeapEntry entry = reduce_warp(thread_best, cmp);
    if (lane == 0) {
      warp_entries[warp] = entry;
    }
    __syncthreads();

    if (warp == 0) {
      HeapEntry block_entry;
      if (lane < BLOCK_WARPS) {
        block_entry = warp_entries[lane];
      } else {
        block_entry = HeapEntry{sentinel, -1, -1, -1};
      }
      block_entry = reduce_warp(block_entry, cmp);
      if (lane == 0) {
        *block_choice = block_entry;
      }
    }
    __syncthreads();

    HeapEntry chosen = *block_choice;
    if (tid == chosen.tid && chosen.slot >= 0) {
      s_vals[base + chosen.slot] = sentinel;
      s_idx[base + chosen.slot] = -1;
    }
    if (tid == 0) {
      size_t out_base = static_cast<size_t>(row) * static_cast<size_t>(k);
      out_vals[out_base + oi] = chosen.value;
      out_idx[out_base + oi] = chosen.column;
    }
    __syncthreads();
  }

  if (tid == 0 && take < k) {
    size_t out_base = static_cast<size_t>(row) * static_cast<size_t>(k);
    for (int oi = take; oi < k; ++oi) {
      out_vals[out_base + oi] = HIP_NAN_F;
      out_idx[out_base + oi] = -1;
    }
    __syncthreads();

    HeapEntry chosen = *block_choice;
    if (tid == chosen.tid && chosen.slot >= 0) {
      s_vals[base + chosen.slot] = sentinel;
      s_idx[base + chosen.slot] = -1;
    }
    if (tid == 0) {
      size_t out_base = static_cast<size_t>(row) * static_cast<size_t>(k);
      out_vals[out_base + oi] = chosen.value;
      out_idx[out_base + oi] = chosen.column;
    }
    __syncthreads();
  }

  if (tid == 0 && take < k) {
    size_t out_base = static_cast<size_t>(row) * static_cast<size_t>(k);
    for (int oi = take; oi < k; ++oi) {
      out_vals[out_base + oi] = HIP_NAN_F;
      out_idx[out_base + oi] = -1;
    }
    __syncthreads();
  }
}

__global__ void topk_warp_heap_rowwise_kernel(
    const float* __restrict__ X, int rows, int cols, int k,
    float* __restrict__ out_vals, int* __restrict__ out_idx)
{
  extern __shared__ unsigned char smem[];
  float* s_vals = (float*)smem;
  int* s_idx = (int*)(s_vals + blockDim.x * KEEP_PER_THREAD);
  __shared__ HeapEntry warp_entries[BLOCK_WARPS];
  __shared__ HeapEntry block_choice;
  heap_select_rowwise_kernel_impl<GreaterThan>(
      X, rows, cols, k, out_vals, out_idx, s_vals, s_idx, warp_entries, &block_choice);
}

__global__ void bottomk_warp_heap_rowwise_kernel(
    const float* __restrict__ X, int rows, int cols, int k,
    float* __restrict__ out_vals, int* __restrict__ out_idx)
{
  extern __shared__ unsigned char smem[];
  float* s_vals = (float*)smem;
  int* s_idx = (int*)(s_vals + blockDim.x * KEEP_PER_THREAD);
  __shared__ HeapEntry warp_entries[BLOCK_WARPS];
  __shared__ HeapEntry block_choice;
  heap_select_rowwise_kernel_impl<LessThan>(
      X, rows, cols, k, out_vals, out_idx, s_vals, s_idx, warp_entries, &block_choice);
}

__global__ void topk_warp_bitonic_rowwise_kernel(
    const float* __restrict__ X, int rows, int cols, int k,
    float* __restrict__ out_vals, int* __restrict__ out_idx)
{
  int row = linear_row_index();
  if (row >= rows) return;
  if (k <= 0) return;

  unsigned mask = __activemask();
  int lane = threadIdx.x & (WARP_LANES - 1);
  const float* row_ptr = X + static_cast<size_t>(row) * static_cast<size_t>(cols);

  if (cols <= 0) {
    if (lane == 0) {
      size_t out_base = static_cast<size_t>(row) * static_cast<size_t>(k);
      for (int oi = 0; oi < k; ++oi) {
        out_vals[out_base + oi] = HIP_NAN_F;
        out_idx[out_base + oi] = -1;
      }
    }
    return;
  }

  float best = -HIP_INF_F;
  int bestc = -1;
  for (int c = lane; c < cols; c += WARP_LANES) {
    float v = row_ptr[c];
    if (v > best || (v == best && (bestc < 0 || c < bestc))) {
      best = v;
      bestc = c;
    }
  }

  for (int offset = WARP_LANES / 2; offset > 0; offset >>= 1) {
    float ov = __shfl_down_sync(mask, best, offset);
    int oc = __shfl_down_sync(mask, bestc, offset);
    if (ov > best || (ov == best && (oc >= 0 && (bestc < 0 || oc < bestc)))) {
      best = ov;
      bestc = oc;
    }
  }

  if (lane == 0) {
    size_t out_base = static_cast<size_t>(row) * static_cast<size_t>(k);
    out_vals[out_base + 0] = best;
    out_idx[out_base + 0] = bestc;
    for (int oi = 1; oi < k; ++oi) {
      out_vals[out_base + oi] = HIP_NAN_F;
      out_idx[out_base + oi] = -1;
    }
  }
}

__global__ void bottomk_warp_bitonic_rowwise_kernel(
    const float* __restrict__ X, int rows, int cols, int k,
    float* __restrict__ out_vals, int* __restrict__ out_idx)
{
  int row = linear_row_index();
  if (row >= rows) return;
  if (k <= 0) return;

  unsigned mask = __activemask();
  int lane = threadIdx.x & (WARP_LANES - 1);
  const float* row_ptr = X + static_cast<size_t>(row) * static_cast<size_t>(cols);

  if (cols <= 0) {
    if (lane == 0) {
      size_t out_base = static_cast<size_t>(row) * static_cast<size_t>(k);
      for (int oi = 0; oi < k; ++oi) {
        out_vals[out_base + oi] = HIP_NAN_F;
        out_idx[out_base + oi] = -1;
      }
    }
    return;
  }

  float best = HIP_INF_F;
  int bestc = -1;
  for (int c = lane; c < cols; c += WARP_LANES) {
    float v = row_ptr[c];
    if (v < best || (v == best && (bestc < 0 || c < bestc))) {
      best = v;
      bestc = c;
    }
  }

  for (int offset = WARP_LANES / 2; offset > 0; offset >>= 1) {
    float ov = __shfl_down_sync(mask, best, offset);
    int oc = __shfl_down_sync(mask, bestc, offset);
    if (ov < best || (ov == best && (oc >= 0 && (bestc < 0 || oc < bestc)))) {
      best = ov;
      bestc = oc;
    }
  }

  if (lane == 0) {
    size_t out_base = static_cast<size_t>(row) * static_cast<size_t>(k);
    out_vals[out_base + 0] = best;
    out_idx[out_base + 0] = bestc;
    for (int oi = 1; oi < k; ++oi) {
      out_vals[out_base + oi] = HIP_NAN_F;
      out_idx[out_base + oi] = -1;
    }
  }
}

} // extern "C"
