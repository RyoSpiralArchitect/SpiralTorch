#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
extern "C" __global__
void hip_compaction_apply_kernel(const float* __restrict__ vin, const int* __restrict__ iin,
                                 int rows, int cols, float low, float high,
                                 const unsigned* __restrict__ flags,
                                 const unsigned* __restrict__ tilecnt, int tiles_per_row,
                                 float* __restrict__ vout, int* __restrict__ iout)
{
    int gid = blockIdx.x;
    int r = gid / tiles_per_row;
    int tile = gid % tiles_per_row;
    if (r >= rows) return;
    int base = r * cols + tile * 256;
    int tid = threadIdx.x;

    unsigned row_off = 0;
    for (int t=0; t<tile; ++t) row_off += tilecnt[r*tiles_per_row + t];

    if (tile*256 + tid < cols) {
        float v = vin[base + tid];
        if (v >= low && v <= high) {
            unsigned pos = flags[base + tid] + row_off;
            vout[r*cols + pos] = v;
            iout[r*cols + pos] = iin[base + tid];
        }
    }
}

extern "C"
hipError_t st_compaction_apply(const float* vin,
                               const int32_t* iin,
                               int rows,
                               int cols,
                               float low,
                               float high,
                               const unsigned* flags,
                               const unsigned* tilecnt,
                               int tiles_per_row,
                               float* vout,
                               int32_t* iout,
                               hipStream_t stream)
{
    if (rows <= 0 || cols <= 0) {
        return hipSuccess;
    }

    if (tiles_per_row <= 0) {
        tiles_per_row = (cols + 255) / 256;
    }

    const int total_tiles = rows * tiles_per_row;
    if (total_tiles <= 0) {
        return hipSuccess;
    }

    dim3 grid(total_tiles);
    dim3 block(256);
    hipLaunchKernelGGL(hip_compaction_apply_kernel, grid, block, 0, stream,
                       vin, iin, rows, cols, low, high, flags, tilecnt, tiles_per_row, vout, iout);
    return hipGetLastError();
}
