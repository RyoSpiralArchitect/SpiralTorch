#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
extern "C" __global__
void hip_compaction_apply_pass(const float* __restrict__ vin,
                               const int*   __restrict__ iin,
                               const unsigned int* __restrict__ pos,
                               int rows, int cols, float low, float high,
                               float* __restrict__ vout,
                               int*   __restrict__ iout)
{
    int r = blockIdx.x; if (r>=rows) return;
    int tid = threadIdx.x;
    int row_off = r*cols;
    for (int c=tid; c<cols; c+=blockDim.x){
        float v = vin[row_off + c];
        if (v>=low && v<=high){
            unsigned p = pos[row_off + c];
            if (p != 0xffffffffu){
                vout[row_off + p] = v;
                iout[row_off + p] = iin[row_off + c];
            }
        }
    }
}
